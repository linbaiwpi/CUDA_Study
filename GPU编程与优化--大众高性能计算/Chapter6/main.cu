#include "ch6.h"

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <hipblas.h>


int main(int argc, char** argv) {
    // CPU memory alloc
    DATATYPE* a = (DATATYPE *)malloc(sizeof(DATATYPE) * N);
    DATATYPE* b = (DATATYPE *)malloc(sizeof(DATATYPE) * N);
    DATATYPE* c = (DATATYPE *)malloc(sizeof(DATATYPE) * N);

    // random data generation
    srand((unsigned int)time(NULL));
    for (int i=0; i<N; ++i) {
        a[i] = ((float)rand() / RAND_MAX) * 100.0f;
        b[i] = ((float)rand() / RAND_MAX) * 100.0f;
    }

    std::cout << "a = ";
    for (int i=0; i<5; ++i) {
        std::cout << a[i] << ", ";
    }
    std::cout << std::endl;
    std::cout << "b = ";
    for (int i=0; i<5; ++i) {
        std::cout << b[i] << ", ";
    }
    std::cout << std::endl;

    std::cout << argv[0]<< std::endl;
    std::cout << argv[1]<< std::endl;

    if (strcmp(argv[1], "6.1") == 0) {
        std::cout << "C reference" << std::endl;
        // c = a + b
        vector_add_serial(a, b, c, N);

        std::cout << "c = ";
        for (int i=0; i<5; ++i) {
            std::cout << c[i] << ", ";
        }
        std::cout << std::endl;

        // CPU memory free
        free(a);
        free(b);
        free(c);        
    } else {
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        float alpha = 1.0;

        std::cout << "GPU reference" << std::endl;
        // GPU memory alloc
        DATATYPE *d_a, *d_b, *d_c;
        hipMalloc((void **) &d_a, sizeof(DATATYPE) * N);
        hipMalloc((void **) &d_b, sizeof(DATATYPE) * N);
        hipMalloc((void **) &d_c, sizeof(DATATYPE) * N);


        if (strcmp(argv[1], "cublas") == 0) {
            hipblasSetVector(N, sizeof(DATATYPE), a, 1, d_a, 1);
            hipblasSetVector(N, sizeof(DATATYPE), b, 1, d_b, 1);
        } else {
            // data a and b copy to GPU
            hipMemcpy(d_a, a, sizeof(DATATYPE) * N, hipMemcpyHostToDevice);
            hipMemcpy(d_b, b, sizeof(DATATYPE) * N, hipMemcpyHostToDevice);
        }

        // c = a + b
        if (strcmp(argv[1], "6.2") == 0) {
            vector_add_gpu_1<<<1,1>>>(d_a, d_b, d_c, N);
        } else if (strcmp(argv[1], "6.3") == 0) {
            int threadnum = 32;
            vector_add_gpu_2<<<1,threadnum>>>(d_a, d_b, d_c, N);
        } else if (strcmp(argv[1], "6.4") == 0) {
            int blocknum = 10;
            int threadnum = 10;
            vector_add_gpu_3<<<blocknum, threadnum>>>(d_a, d_b, d_c, N);
        } else if (strcmp(argv[1], "cublas") == 0) {
            std::cout << "Calling cublas" << std::endl;
            hipblasSaxpy(handle, N, &alpha, d_a, 1, d_b, 1);
        } else {
            std::cout << "" << std::endl;
        }

        // result copy back to CPU
        if (strcmp(argv[1], "cublas") == 0) {
            hipblasGetVector(N, sizeof(DATATYPE), d_b, 1, c, 1);
        } else {
            hipMemcpy(c, d_c, sizeof(DATATYPE) * N, hipMemcpyDeviceToHost);
        }

        std::cout << "c = ";
        for (int i=0; i<5; ++i) {
            std::cout << c[i] << ", ";
        }
        std::cout << std::endl;

        // GPU memory free
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);

        hipblasDestroy(handle);
    }
}
